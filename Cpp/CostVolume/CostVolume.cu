#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#define cudaSafeCall(expr)  __cudaSafeCall(expr, __FILE__, __LINE__, __func__)

void __cudaSafeCall(hipError_t err, const char* file, const int line, const char* func = "")
{
    if (hipSuccess != err)
        std::cout<<hipGetErrorString(err)<<"\nFile:"<<file<<":"<< line<< " IN "<<func<<std::endl;
}
namespace cv { namespace gpu { namespace device {
    namespace dtam_updateCost{

    struct m33{
        float data [9];
    };
//__constant__ float sliceToIm[3 * 3];
__constant__ uint  rows;
__constant__ uint  cols;
__constant__ uint  layers;
__constant__ uint layerStep;
__constant__ float* hdata;
__constant__ float* cdata;
__constant__ float* lo;
__constant__ float* hi;
__constant__ uint* loInd;
__constant__ float3* base;
__constant__ hipTextureObject_t tex;

__global__ void updateCostCol(m33 sliceToIm, unsigned int yoff);

#define SEND(type,sym) hipMemcpyToSymbol(HIP_SYMBOL(sym), &h_ ## sym, sizeof(type));

void loadConstants(int h_layers, int h_layerStep, float3* h_base,
        float* h_hdata, float* h_cdata, float* h_lo, float* h_hi, uint* h_loInd,
        uint h_rows, uint h_cols, hipTextureObject_t h_tex) {
//    hipMemcpyToSymbol(HIP_SYMBOL(c_sliceToIm), h_CVToIm, 3*4*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(layers), &h_layers, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(layerStep), &h_layerStep, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(hdata), &h_hdata, sizeof(float*));
    hipMemcpyToSymbol(HIP_SYMBOL(cdata), &h_cdata, sizeof(float*));
    assert(h_lo);
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lo), &h_lo, sizeof(float*)));
    hipMemcpyToSymbol(HIP_SYMBOL(hi), &h_hi, sizeof(float*));
    hipMemcpyToSymbol(HIP_SYMBOL(loInd), &h_loInd, sizeof(uint*));
    SEND(uint,cols);
    SEND(uint,rows);
    SEND(float3*,base);
    SEND(hipTextureObject_t,tex);
}

//void loadS2I(float* h_sliceToIm){
//    hipMemcpyToSymbol(HIP_SYMBOL(sliceToIm), h_sliceToIm, 3*3*sizeof(float));
//}


void updateCostColCaller(int cols,int rows, int y, m33 sliceToIm){
    static hipStream_t strs [15]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    static int num=0;
    int here=(num++%14)+1;
    if(!strs[here])
        hipStreamCreate(&(strs[here]));
   dim3 dimBlock(64,1);
   dim3 dimGrid((cols  + dimBlock.x - 1) / dimBlock.x,
                (rows + dimBlock.y - 1) / dimBlock.y);
   updateCostCol<<<dimGrid, dimBlock,1,strs[here]>>>(sliceToIm,y*cols);
}

__global__ void updateCostCol(m33 c_sliceToIm,unsigned int yoff)
{
    //per thread:
        //find projection from column to image (3x2)
        //for each pixel:
            //finds L1 error
            //blend in with old value
            //if low
                //update low index
                //update high value
            //if high
                //update high value
        //save results
    //    x  y  z  1 
    // x  0     1  2
    // y  3     4  5
    // z  6     7  8

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int offset=x+yoff;
    
    
    // Find the current base color
    float3 B = base[offset];

    float3 Z;
    float3 W;
    Z.x = c_sliceToIm.data[1];
    Z.y = c_sliceToIm.data[4];
    Z.z = c_sliceToIm.data[7];

    //add in the offset for the current column
    W.x = c_sliceToIm.data[2] + c_sliceToIm.data[0] * x;
    W.y = c_sliceToIm.data[5] + c_sliceToIm.data[3] * x;
    W.z = c_sliceToIm.data[8] + c_sliceToIm.data[6] * x;
    float *hp = hdata+offset;
    float *cp = cdata+offset;
    float minv = 1000;
    float maxv = 0;
    uint mini=0;
    uint end=layers*layerStep;
    for (uint z = 0; z < end; z+=layerStep, W.x += Z.x, W.y += Z.y, W.z += Z.z) {
        //Texture Sample
            float xi, yi, wi;
            wi = W.z;
            xi = W.x / wi;
            yi = W.y / wi;
            float4 c = tex2D<float4>(tex, xi, yi);

        {//Shader core
            float v1 = fabsf(c.x - B.x);
            float v2 = fabsf(c.y - B.y);
            float v3 = fabsf(c.z - B.z);
            float h = hp[z] + 1;
            float ns = cp[z] * (1 - 1 / h) + (v1 + v2 + v3) / h;
            if (ns < minv) {
                minv = ns;
                mini = z;
            }
            if (ns > maxv) {
                maxv = ns;
            }

            hp[z] = h;
            cp[z] = ns;
        }
    }
    lo[offset]=minv;
    hi[offset]=maxv;
    loInd[offset]=mini;
}

}}}}




